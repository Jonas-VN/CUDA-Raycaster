#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <stdio.h>
#include "settings.hpp"
#include "player.h"
#include "map.hpp"
#include <iostream>
#include <string>

#if TEST_MEMORY_LEAKS
#include <crtdbg.h>
#endif

#if USE_GPU
#include <hip/hip_runtime.h>
#include <>

__constant__ int gpuMap[MAP_HEIGHT][MAP_WIDTH];
#endif

#if USE_TEXTURE
#include <SDL_image.h>
#endif

SDL_Window* gWindow = nullptr;
SDL_Surface* gSurface = nullptr;

// This may be ugly code, but it does show the difference between GPU and CPU code rather easily without much effort
#if USE_GPU && USE_TEXTURE && USE_TEXTURE_OBJECT
__global__ void raycast(Uint32* pixels, Player* player, hipTextureObject_t texture) {
#elif USE_GPU && USE_TEXTURE && !USE_TEXTURE_OBJECT
__global__ void raycast(Uint32 * pixels, Player * player, Uint32* texture) {
#elif USE_GPU && !USE_TEXTURE
__global__ void raycast(Uint32 * pixels, Player * player) {
#elif !USE_GPU && USE_TEXTURE
void raycast(Uint32 * pixels, Player * player, const Map * map, Uint32 * texture) {
#elif !USE_GPU && !USE_TEXTURE
void raycast(Uint32 * pixels, Player * player, const Map * map) {
#endif

// On the GPU: if statment to check bounds
// On the CPU: for loop
#if USE_GPU
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    if (column < SCREEN_WIDTH) {
#else
    for (int column = 0; column < SCREEN_WIDTH; column++) {
#endif
        // Calculate ray direction
        double factor = -1.0 + 2.0 * column / SCREEN_WIDTH;
        Direction rayDirection = Direction(
            player->camera->distanceToPlayer * player->direction->x + factor * player->camera->direction->x,
            player->camera->distanceToPlayer * player->direction->y + factor * player->camera->direction->y
        );
        rayDirection.normalize();

        // Raycast
        int x = 0;
        int y = 0;
        double delta_v = abs(1 / rayDirection.x);
        double delta_h = abs(1 / rayDirection.y);

        double verticalDistance;
        if (rayDirection.x < 0) verticalDistance = (player->coordinate->x - floor(player->coordinate->x)) * delta_v;
        else verticalDistance = (ceil(player->coordinate->x) - player->coordinate->x) * delta_v;

        double horizontalDistance;
        if (rayDirection.y < 0) horizontalDistance = (player->coordinate->y - floor(player->coordinate->y)) * delta_h;
        else horizontalDistance = (ceil(player->coordinate->y) - player->coordinate->y) * delta_h;

        Coordinate roundedIntersection;
        Coordinate intersection;
        int hitDirection = 0;
        double distanceToWall = 0.0;
        bool hit = false;
        while (!hit) {
            if (verticalDistance + y * delta_v < horizontalDistance + x * delta_h) {
                factor = verticalDistance + y * delta_v;
                intersection = Coordinate(
                    rayDirection.x * factor + player->coordinate->x,
                    rayDirection.y * factor + player->coordinate->y
                );

                if (rayDirection.x < 0) {
                    roundedIntersection = Coordinate(
                        round(intersection.x) - 1.0,
                        floor(intersection.y)
                    );
                }
                else {
                    roundedIntersection = Coordinate(
                        round(intersection.x),
                        floor(intersection.y)
                    );
                }
#if USE_GPU
                if (gpuMap[(int)roundedIntersection.y][(int)roundedIntersection.x]) {
#else
                if (map->isWall(&roundedIntersection)) {
#endif
                    hit = true;
                    distanceToWall = factor * rayDirection.dotProduct(player->direction);
                    hitDirection = 1;
                }
                y++;
                }
            else {
                factor = horizontalDistance + x * delta_h;
                intersection = Coordinate(
                    rayDirection.x * factor + player->coordinate->x,
                    rayDirection.y * factor + player->coordinate->y
                );
                if (rayDirection.y < 0) {
                    roundedIntersection = Coordinate(
                        floor(intersection.x),
                        round(intersection.y) - 1.0
                    );
                }
                else {
                    roundedIntersection = Coordinate(
                        floor(intersection.x),
                        round(intersection.y)
                    );
                }
#if USE_GPU
                if (gpuMap[(int)roundedIntersection.y][(int)roundedIntersection.x]) {
#else
                if (map->isWall(&roundedIntersection)) {
#endif
                    hit = true;
                    distanceToWall = factor * rayDirection.dotProduct(player->direction);
                }
                x++;
                }
            }

        double length = 1 / distanceToWall * SCREEN_HEIGHT;
        int start = (SCREEN_HEIGHT - length) / 2;
        int end = start + length;

        int realStart = start >= 0 ? start : 0;
        int realEnd = end <= SCREEN_HEIGHT ? end : SCREEN_HEIGHT;

#if USE_TEXTURE
        int textureX;
        if (hitDirection == 0) textureX = (int)((intersection.x - floor(intersection.x)) * TEXTURE_WIDTH);
        else textureX = (int)((intersection.y - floor(intersection.y)) * TEXTURE_WIDTH);
        double ratio = (double)TEXTURE_HEIGHT / (end - start);
        double textureY = start >= 0 ? 0.0 : -start;
#endif

        for (int y = realStart; y < realEnd; ++y) {
#if USE_TEXTURE
            int sourceY = (int)(textureY++ * ratio);
#endif
#if USE_GPU && USE_TEXTURE && USE_TEXTURE_OBJECT
            pixels[y * SCREEN_WIDTH + column] = tex1Dfetch<Uint32>(texture, sourceY * TEXTURE_WIDTH + textureX);
#elif USE_GPU && USE_TEXTURE && !USE_TEXTURE_OBJECT
            pixels[y * SCREEN_WIDTH + column] = texture[sourceY * TEXTURE_WIDTH + textureX];
#elif !USE_GPU && USE_TEXTURE
            pixels[y * SCREEN_WIDTH + column] = texture[sourceY * TEXTURE_WIDTH + textureX];
#else
            pixels[y * SCREEN_WIDTH + column] = 0xFF0000 - hitDirection * 0x330000;
#endif
        }
    }
}

bool initSDL() {
    // Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        printf("SDL could not initialize! SDL_Error: %s\n", SDL_GetError());
        return false;
    }

#if USE_TEXTURE
    // Initialize SDL_image
    int imgFlags = IMG_INIT_PNG;
    if ((IMG_Init(imgFlags) & imgFlags) != imgFlags) {
        std::cerr << "SDL_image initialization failed: " << IMG_GetError() << std::endl;
        SDL_Quit();
        return false;
    }
#endif

    // Create window
    gWindow = SDL_CreateWindow("Raycaster (FPS: )", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (gWindow == nullptr) {
        printf("Window could not be created! SDL_Error: %s\n", SDL_GetError());
        return false;
    }

    // Get window surface
    gSurface = SDL_GetWindowSurface(gWindow);
    SDL_SetRelativeMouseMode(SDL_TRUE);

    return true;
}

void closeSDL() {
    // Destroy window
    SDL_DestroyWindow(gWindow);
    gWindow = nullptr;

#if USE_TEXTURE
    // Quit SDL_image
    IMG_Quit();
#endif

    // Quit SDL subsystems
    SDL_Quit();
}

#if USE_TEXTURE
Uint32* loadImage(const char* filepath) {
    SDL_Surface* surface = IMG_Load(filepath);
    surface = SDL_ConvertSurfaceFormat(surface, SDL_PIXELFORMAT_RGBA32, 0);
    if (!surface) {
        std::cerr << "Failed to load image: " << IMG_GetError() << std::endl;
        IMG_Quit();
        SDL_Quit();
        return false;
    }
    return static_cast<Uint32*>(surface->pixels);
}
#endif

bool handle_keys(double delta, Player * player) {
    SDL_Event e;
    bool quit = false;

    while (SDL_PollEvent(&e) != 0) {
        if (e.type == SDL_QUIT) {
            quit = true;
        }
        else if (e.type == SDL_MOUSEMOTION) {
            player->rotate(e.motion.xrel);
        }
    }

    const Uint8* keys = SDL_GetKeyboardState(NULL);
    if (keys[SDL_SCANCODE_ESCAPE]) {
        quit = true;
    }
    if (keys[SDL_SCANCODE_W]) {
        player->moveY(delta);
    }
    if (keys[SDL_SCANCODE_S]) {
        player->moveY(-delta);
    }
    if (keys[SDL_SCANCODE_D]) {
        player->moveX(delta);
    }
    if (keys[SDL_SCANCODE_A]) {
        player->moveX(-delta);
    }

    return quit;
}


int main(int argc, char* args[]) {
    // Initialize SDL
    if (!initSDL()) {
        printf("Failed to initialize!\n");
        return -1;
    }

    const Map* map = new Map();
    Player* player = new Player(map);

    double totalDeltaTime = 0.0;
    int numFrames = 0;

#if USE_TEXTURE
    Uint32* texture = loadImage("src/wall.png");
#endif

#if USE_GPU
    hipMemcpyToSymbol(HIP_SYMBOL(gpuMap), map, MAP_HEIGHT * MAP_WIDTH * sizeof(int));

    Uint32* gpuPixels;
    Player* gpuPlayer;
    Camera* gpuCamera;
    Direction* gpuPlayerDirection;
    Coordinate* gpuPlayerCoordinate;
    Direction* gpuCameraDirection;
    hipMalloc((void**)&gpuPixels, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(Uint32));
    hipMalloc((void**)&gpuPlayer, sizeof(Player));
    hipMalloc((void**)&gpuCamera, sizeof(Camera));
    hipMalloc((void**)&gpuPlayerDirection, sizeof(Direction));
    hipMalloc((void**)&gpuPlayerCoordinate, sizeof(Coordinate));
    hipMalloc((void**)&gpuCameraDirection, sizeof(Direction));

    // The player & camera are technically constant since only the pointers are relevant or the other fields are constant
    hipMemcpy(gpuPlayer, player, sizeof(Player), hipMemcpyHostToDevice);
    hipMemcpy(gpuCamera, player->camera, sizeof(Camera), hipMemcpyHostToDevice);

    // Populate pointers (these can be copied in advance because the gpuPlayer & gpuCamera never changes and thus the pointers stay relevant)
    hipMemcpy(&(gpuPlayer->direction), &gpuPlayerDirection, sizeof(Direction*), hipMemcpyHostToDevice);
    hipMemcpy(&(gpuPlayer->coordinate), &gpuPlayerCoordinate, sizeof(Coordinate*), hipMemcpyHostToDevice);
    hipMemcpy(&(gpuPlayer->camera), &gpuCamera, sizeof(Camera*), hipMemcpyHostToDevice);
    hipMemcpy(&(gpuCamera->direction), &gpuCameraDirection, sizeof(Direction*), hipMemcpyHostToDevice);

    int blockSize = 128;
    int numBlocks = (SCREEN_WIDTH + blockSize - 1) / blockSize;
#endif

#if USE_GPU && USE_TEXTURE
    Uint32* textureData;
    hipMalloc((void**)&textureData, TEXTURE_HEIGHT * TEXTURE_WIDTH * sizeof(Uint32));
    hipMemcpy(textureData, texture, TEXTURE_HEIGHT * TEXTURE_WIDTH * sizeof(Uint32), hipMemcpyHostToDevice);
#endif

#if USE_GPU && USE_TEXTURE && USE_TEXTURE_OBJECT
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = textureData;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = TEXTURE_HEIGHT * TEXTURE_WIDTH * sizeof(Uint32);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t gpuTexture = 0;
    hipCreateTextureObject(&gpuTexture, &resDesc, &texDesc, NULL);
#endif

    double currentTime = (double)SDL_GetTicks64();
    double prevTime = currentTime;
    bool quit = false;
    while (!quit) {
        currentTime = (double)SDL_GetTicks64();
        double delta = (currentTime - prevTime) / 1000.0;
        totalDeltaTime += delta;
        numFrames++;
        prevTime = currentTime;
        int fps = (int)(1 / delta);

        if (SDL_LockSurface(gSurface) == 0) {
            Uint32* pixels = (Uint32*)gSurface->pixels;

#if USE_GPU
            hipMemset(gpuPixels, BACKGROUND_COLOR, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(Uint32));
            // Copy the content of the player
            hipMemcpy(gpuPlayerDirection, player->direction, sizeof(Direction), hipMemcpyHostToDevice);
            hipMemcpy(gpuPlayerCoordinate, player->coordinate, sizeof(Coordinate), hipMemcpyHostToDevice);
            hipMemcpy(gpuCameraDirection, player->camera->direction, sizeof(Direction), hipMemcpyHostToDevice);
#else
            memset(pixels, BACKGROUND_COLOR, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(Uint32));
#endif

#if USE_GPU && USE_TEXTURE && USE_TEXTURE_OBJECT
            raycast << <numBlocks, blockSize >> > (gpuPixels, gpuPlayer, gpuTexture);
#elif USE_GPU && USE_TEXTURE && !USE_TEXTURE_OBJECT
            raycast << <numBlocks, blockSize >> > (gpuPixels, gpuPlayer, textureData);
#elif USE_GPU && !USE_TEXTURE
            raycast << <numBlocks, blockSize >> > (gpuPixels, gpuPlayer);
#elif !USE_GPU && USE_TEXTURE
            raycast(pixels, player, map, texture);
#elif !USE_GPU && !USE_TEXTURE
            raycast(pixels, player, map);
#endif

            quit = handle_keys(delta, player);

#if USE_GPU
            hipDeviceSynchronize();
            hipMemcpy(pixels, gpuPixels, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(Uint32), hipMemcpyDeviceToHost);
#endif

            gSurface->pixels = pixels;
            SDL_UnlockSurface(gSurface);
            SDL_UpdateWindowSurface(gWindow);
            std::string windowTitle = "Raycaster (FPS: " + std::to_string(fps) + ")";
            SDL_SetWindowTitle(gWindow, windowTitle.c_str());

#if BENCHMARKING
            if (totalDeltaTime >= 10) quit = true;
#endif
        }
    }

    closeSDL();

    delete(player);
    delete(map);

#if USE_GPU
    hipFree(gpuPixels);
    hipFree(gpuPlayer);
    hipFree(gpuPlayerDirection);
    hipFree(gpuPlayerCoordinate);
    hipFree(gpuCameraDirection);
    hipFree(gpuCamera);
#endif

#if USE_GPU && USE_TEXTURE && USE_TEXTURE_OBJECT
    hipDestroyTextureObject(gpuTexture);
#endif

#if USE_GPU && USE_TEXTURE
    hipFree(textureData);
#endif

    std::cout << "Average FPS: " << 1 / (totalDeltaTime / numFrames) << std::endl;

#if TEST_MEMORY_LEAKS
    _CrtDumpMemoryLeaks();
#endif

    return 0;
}