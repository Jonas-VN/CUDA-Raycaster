#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <stdio.h>
#include "settings.hpp"
#include "player.h"
#include "map.hpp"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <>


SDL_Window* gWindow = nullptr;
SDL_Surface* gSurface = nullptr;


#if USE_GPU
__global__ void GPU_Raycast(Uint32* pixels, double playerDirectionX, double playerDirectionY, double playerCoordinateX, double playerCoordinateY, double cameraDirectionX, double cameraDirectionY, double cameraDistance) {
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    if (column < SCREEN_WIDTH) {
        // Calculate ray direction
        double factor = -1.0 + 2.0 * column / SCREEN_WIDTH;
        double rayDirectionX = cameraDistance * playerDirectionX + factor * cameraDirectionX;
        double rayDirectionY = cameraDistance * playerDirectionY + factor * cameraDirectionY;
        double magnitude = sqrt(rayDirectionX * rayDirectionX + rayDirectionY * rayDirectionY);
        if (magnitude != 0) {
            rayDirectionX /= magnitude;
            rayDirectionY /= magnitude;
        }

        // Raycast
        int x = 0;
        int y = 0;
        double delta_v = abs(1 / rayDirectionX);
        double delta_h = abs(1 / rayDirectionY);

        double verticalDistance;
        if (rayDirectionX < 0) verticalDistance = (playerCoordinateX - floor(playerCoordinateX)) * delta_v;
        else verticalDistance = (ceil(playerCoordinateX) - playerCoordinateX) * delta_v;

        double horizontalDistance;
        if (rayDirectionY < 0) horizontalDistance = (playerCoordinateY - floor(playerCoordinateY)) * delta_h;
        else horizontalDistance = (ceil(playerCoordinateY) - playerCoordinateY) * delta_h;

        double intersectionX;
        double intersectionY;
        int hitDirection = 0;
        double distanceToWall = 0.0;
        bool hit = false;
        while (!hit) {
            if (verticalDistance + y * delta_v < horizontalDistance + x * delta_h) {
                factor = verticalDistance + y * delta_v;
                if (rayDirectionX < 0) {
                    intersectionX = round(rayDirectionX * factor + playerCoordinateX) - 1.0;
                    intersectionY = floor(rayDirectionY * factor + playerCoordinateY);
                }
                else {
                    intersectionX = round(rayDirectionX * factor + playerCoordinateX);
                    intersectionY = floor(rayDirectionY * factor + playerCoordinateY);
                }
                if (d_map[(int) intersectionY][(int) intersectionX] == 1) {
                    hit = true;
                    distanceToWall = factor * (rayDirectionX * playerDirectionX + rayDirectionY * playerDirectionY);
                    hitDirection = 1;
                }
                y++;
            }
            else {
                factor = horizontalDistance + x * delta_h;
                if (rayDirectionY < 0) {
                    intersectionX = floor(rayDirectionX * factor + playerCoordinateX);
                    intersectionY = round(rayDirectionY * factor + playerCoordinateY) - 1.0;
                }
                else {
                    intersectionX = floor(rayDirectionX * factor + playerCoordinateX);
                    intersectionY = round(rayDirectionY * factor + playerCoordinateY);
                }

                if (d_map[(int)intersectionY][(int)intersectionX] == 1) {
                    hit = true;
                    distanceToWall = factor * (rayDirectionX * playerDirectionX + rayDirectionY * playerDirectionY);
                }
                x++;
            }
        }

        double length = 1 / distanceToWall * SCREEN_HEIGHT;
        int start = (SCREEN_HEIGHT - length) / 2 >= 0 ? (int)(SCREEN_HEIGHT - length) / 2 : 0;
        int end = start + length <= SCREEN_HEIGHT ? (int)start + length : SCREEN_HEIGHT;
        for (int y = start; y < end; ++y) {
            pixels[y * SCREEN_WIDTH + column] = hitDirection == 0 ? 0xFF0000 : 0xDD0000;
        }
    }
}
#else
void CPU_Raycast(Uint32* pixels, Player* player, const Map* map) {
    double factor;
    for (int column = 0; column < SCREEN_WIDTH; column++) {
        // Calculate ray direction
        factor = -1.0 + 2.0 * column / SCREEN_WIDTH;
        Direction rayDirection = Direction(
            player->camera->distanceToPlayer * player->direction->x + factor * player->camera->direction->x,
            player->camera->distanceToPlayer * player->direction->y + factor * player->camera->direction->y
        );
        rayDirection.normalize();

        // Raycast
        int x = 0;
        int y = 0;
        double delta_v = abs(1 / rayDirection.x);
        double delta_h = abs(1 / rayDirection.y);

        double verticalDistance;
        if (rayDirection.x < 0) verticalDistance = (player->coordinate->x - std::floor(player->coordinate->x)) * delta_v;
        else verticalDistance = (std::ceil(player->coordinate->x) - player->coordinate->x) * delta_v;

        double horizontalDistance;
        if (rayDirection.y < 0) horizontalDistance = (player->coordinate->y - std::floor(player->coordinate->y)) * delta_h;
        else horizontalDistance = (std::ceil(player->coordinate->y) - player->coordinate->y) * delta_h;

        Coordinate intersection;
        int hitDirection = 0;
        double distanceToWall = 0.0;
        bool hit = false;
        while (!hit) {
            if (verticalDistance + y * delta_v < horizontalDistance + x * delta_h) {
                factor = verticalDistance + y * delta_v;
                if (rayDirection.x < 0) {
                    intersection = Coordinate(
                        std::round(rayDirection.x * factor + player->coordinate->x) - 1.0,
                        std::floor(rayDirection.y * factor + player->coordinate->y)
                    );
                }
                else {
                    intersection = Coordinate(
                        std::round(rayDirection.x * factor + player->coordinate->x),
                        std::floor(rayDirection.y * factor + player->coordinate->y)
                    );
                }
                if (map->isWall(&intersection)) {
                    hit = true;
                    distanceToWall = factor * rayDirection.dotProduct(player->direction);
                    hitDirection = 1;
                }
                y++;
            }
            else {
                factor = horizontalDistance + x * delta_h;
                if (rayDirection.y < 0) {
                    intersection = Coordinate(
                        std::floor(rayDirection.x * factor + player->coordinate->x),
                        std::round(rayDirection.y * factor + player->coordinate->y) - 1.0
                    );
                }
                else {
                    intersection = Coordinate(
                        std::floor(rayDirection.x * factor + player->coordinate->x),
                        std::round(rayDirection.y * factor + player->coordinate->y)
                    );
                }

                if (map->isWall(&intersection)) {
                    hit = true;
                    distanceToWall = factor * rayDirection.dotProduct(player->direction);
                }
                x++;
            }
        }

        double length = 1 / distanceToWall * SCREEN_HEIGHT;
        int start = (SCREEN_HEIGHT - length) / 2 >= 0 ? (int)(SCREEN_HEIGHT - length) / 2 : 0;
        int end = start + length <= SCREEN_HEIGHT ? (int)start + length : SCREEN_HEIGHT;

        for (int y = start; y < end; ++y) {
            pixels[y * SCREEN_WIDTH + column] = SDL_MapRGB(gSurface->format, 255 - hitDirection * 100, 0, 0);
        }
    }
}
#endif


bool initSDL() {
    // Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        printf("SDL could not initialize! SDL_Error: %s\n", SDL_GetError());
        return false;
    }

    // Create window
    gWindow = SDL_CreateWindow("Raycaster (FPS: )", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (gWindow == nullptr) {
        printf("Window could not be created! SDL_Error: %s\n", SDL_GetError());
        return false;
    }

    // Get window surface
    gSurface = SDL_GetWindowSurface(gWindow);
    SDL_SetRelativeMouseMode(SDL_TRUE);

    return true;
}

void closeSDL() {
    // Destroy window
    SDL_DestroyWindow(gWindow);
    gWindow = nullptr;

    // Quit SDL subsystems
    SDL_Quit();
}

bool handle_keys(double delta, Player* player) {
    SDL_Event e;
    bool quit = false;

    while (SDL_PollEvent(&e) != 0) {
        if (e.type == SDL_QUIT) {
            quit = true;
        }
        else if (e.type == SDL_MOUSEMOTION) {
            player->rotate(e.motion.xrel);
        }
    }

    const Uint8* keys = SDL_GetKeyboardState(NULL);
    if (keys[SDL_SCANCODE_ESCAPE]) {
        quit = true;
    }
    if (keys[SDL_SCANCODE_W]) {
        player->moveY(delta);
    }
    if (keys[SDL_SCANCODE_S]) {
        player->moveY(-delta);
    }
    if (keys[SDL_SCANCODE_D]) {
        player->moveX(delta);
    }
    if (keys[SDL_SCANCODE_A]) {
        player->moveX(-delta);
    }

    return quit;
}


int main(int argc, char* args[]) {
    // Initialize SDL
    if (!initSDL()) {
        printf("Failed to initialize!\n");
        return -1;
    }

    const Map* map = new Map();
    Player* player = new Player(map);

#if USE_GPU
    map->copyMapToGPU();
    Uint32* gpuPixels;
    hipMalloc((void**)&gpuPixels, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(Uint32));
    int blockSize = 256;
    int numBlocks = (SCREEN_WIDTH + blockSize - 1) / blockSize;
    std::cout << numBlocks << std::endl;
#endif

    double currentTime = SDL_GetTicks64();
    double prevTime = currentTime;
    bool quit = false;
    while (!quit) {
        currentTime = SDL_GetTicks64();
        double delta = (currentTime - prevTime) / 1000;
        prevTime = currentTime;
        int fps = (int) 1 / delta;

        if (SDL_LockSurface(gSurface) == 0) {
            Uint32* pixels = (Uint32*)gSurface->pixels;

#if USE_GPU
            hipMemset(gpuPixels, 0x000000, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(Uint32));
            GPU_Raycast << <numBlocks, blockSize >> > (gpuPixels, player->direction->x, player->direction->y, player->coordinate->x, player->coordinate->y, player->camera->direction->x, player->camera->direction->y, player->camera->distanceToPlayer);
            quit = handle_keys(delta, player);
            hipDeviceSynchronize();
            hipMemcpy(pixels, gpuPixels, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(Uint32), hipMemcpyDeviceToHost);
#else
            memset(pixels, 0x000000, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(Uint32));
            CPU_Raycast(pixels, player, map);
            quit = handle_keys(delta, player);
#endif

            gSurface->pixels = pixels;
            SDL_UnlockSurface(gSurface);
            SDL_UpdateWindowSurface(gWindow);
            std::string windowTitle = "Raycaster (FPS: " + std::to_string(fps) + ")";
            SDL_SetWindowTitle(gWindow, windowTitle.c_str());
        }
    }

    closeSDL();

    delete(player);
    delete(map);

#if USE_GPU
    hipFree(gpuPixels);
#endif
    return 0;
}



